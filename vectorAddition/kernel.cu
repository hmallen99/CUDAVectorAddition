#include <iostream>
#include "hip/hip_runtime.h"


#include <stdio.h>

__global__ void add(int *a, int *b, int *c) {
	c[blockIdx.x] = a[blockIdx.x] + b[blockIdx.x];
}

void random_ints(int* a, int N) {
	for (int i = 0; i < N; i++) {
		a[i] = rand() % 10;
	}
}

#define N 10
int main(void)
{
	int *a, *b, *c;
	int *d_a, *d_b, *d_c;
	int size = N * sizeof(int);

	hipMalloc((void **)&d_a, size);
	hipMalloc((void **)&d_b, size);
	hipMalloc((void **)&d_c, size);

	a = (int *)malloc(size); random_ints(a, N);
	b = (int *)malloc(size); random_ints(b, N);
	c = (int *)malloc(size);

	hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
	hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);

	add<<<N, 1>>> (d_a, d_b, d_c);

	hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);

	for (int i = 0; i < N; i++) {
		std::cout << a[i] << " + " << b[i] << " = " << c[i] << std::endl;
	}

	free(a); free(b); free(c);
	hipFree(d_a); hipFree(d_c); hipFree(d_c);
	
    return 0;
}

